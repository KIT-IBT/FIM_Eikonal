#include "hip/hip_runtime.h"
//
//  main.cu
//  tetFIM
//
//  Created by Steffen Schuler in July 2020.
//  Copyright © 2020 IBT. All rights reserved.
//

#include <Eikonal.h>
#include <vtkSmartPointer.h>
#include <vtkDataSetReader.h>
#include <vtkXMLUnstructuredGridReader.h>
#include <vtkUnstructuredGrid.h>
#include <vtkCell.h>
#include <vtkCellData.h>
#include <sstream>
#include <fstream>
#include <iostream>
#include <string>
#include <cmath>
#include "Matrix3.h"

Matrix3<float> calculateTensor(float phi, float theta, float anisotropy)
{
    // angles are defined according to the ISO convention for rotational coordinates, see
    // https://commons.wikimedia.org/w/index.php?title=File:Kugelkoord-def.svg&oldid=378536762
    // the positive z-axis (element 8 of result) represents the initial fiber direction before rotation
    
    float cp = cos(phi);
    float sp = sin(phi);
    Matrix3<float> rotZ(cp, -sp, 0,
                        sp,  cp, 0,
                         0,   0, 1);
    
    float ct = cos(theta);
    float st = sin(theta);
    Matrix3<float> rotY( ct, 0, st,
                          0, 1,  0,
                        -st, 0, ct);
    
    Matrix3<float> rot = rotZ * rotY;
    
    Matrix3<float> result;
    result.SetToIdentityMatrix();
    result(8) = 1.0 / (anisotropy * anisotropy);
    result = rot * result * rot.GetTranspose();
    
    return result;
}

int main(int argc, char *argv[])
{
    std::string inFile;
    std::string seedsFile;
    std::string outFile;
    bool verbose = false;
    float anisotropy = 1.0;
    bool anisotropyProvided = false;
    float speed = 1.0;
    bool speedProvided = false;
    int maxBlocks = 16;
    int maxVertsPerBlock = 24;
    int maxIterations = 2000;
    
    for(int i = 0; i < argc; i++)
    {
        if(argc < 2 || strcmp(argv[i], "-help") == 0 || strcmp(argv[i], "-h") == 0)
        {
            std::cout << std::endl;
            std::cout << "Syntax:" << std::endl << std::endl;
            std::cout << "  tetFIM -i inFile.vtu -p seedsFile.txt -o outFile.txt {additional options}" << std::endl;
            std::cout << std::endl;
            std::cout << "Available options:" << std::endl << std::endl;
            std::cout << "  -i  or -inFile              VTK unstructured grid dataset (.vtu or .vtk)" << std::endl;
            std::cout << "  -p  or -seedsFile           .txt" << std::endl;
            std::cout << "  -o  or -outFile             .bin or .txt" << std::endl;
            std::cout << "  -a  or -anisotropy          default: 1.0" << std::endl;
            std::cout << "  -s  or -speed               default: 1.0" << std::endl;
            std::cout << "  -m  or -maxIterations       default: 2000" << std::endl;
            std::cout << "  -b  or -maxBlocks           default: 16" << std::endl;
            std::cout << "  -vb or -maxVertsPerBlock    default: 24" << std::endl;
            std::cout << "  -v  or -verbose" << std::endl;
            std::cout << "  -h  or -help" << std::endl;
            std::cout << std::endl;
            std::cout << "Hints:" << std::endl << std::endl;
            std::cout << "  The seeds file is used to provide IDs of starting points (value zero in the solution)." << std::endl;
            std::cout << "  It may consist of multiple lines defining different 'seed configurations', each of which is solved individually." << std::endl;
            std::cout << "  Within each line, point IDs have to be separated by a space." << std::endl;
            std::cout << std::endl;
            std::cout << "  If the output file has the extension '.bin', the output will be binary, otherwise ascii." << std::endl;
            std::cout << "  For binary, the first value will be the number of points as int32, followed by the values for each seed configuration as float." << std::endl;
            std::cout << "  For ascii, there will be one column for each point in the mesh and one row for each seed configuration." << std::endl;
            std::cout << std::endl;
            std::cout << "  Cell data arrays 'Phi' and 'Theta' containing angles in radian are required in the VTK dataset for anisotropy != 1." << std::endl;
            std::cout << std::endl;
            std::cout << "  A cell data array 'Speed' may be provided in the VTK dataset to define different speeds for each tetrahedron." << std::endl;
            std::cout << std::endl;
            std::cout << "  Try to reduce maxVertsPerBlock if you get 'cudaCheckError() : invalid configuration argument'." << std::endl;
            std::cout << "  This parameter determines maxNumTotalTets, which must be smaller than the max number of threads per block for your CUDA device (usually 512 or 1024)." << std::endl;
            std::cout << std::endl;
            return(EXIT_SUCCESS);
        }
        else if((strcmp(argv[i], "-inFile") == 0 || strcmp(argv[i], "-i") == 0) && argc > i+1)
            inFile = argv[i+1];
        else if((strcmp(argv[i], "-seedsFile") == 0 || strcmp(argv[i], "-p") == 0) && argc > i+1)
            seedsFile = argv[i+1];
        else if((strcmp(argv[i], "-outFile") == 0 || strcmp(argv[i], "-o") == 0) && argc > i+1)
            outFile = argv[i+1];
        else if((strcmp(argv[i], "-anisotropy") == 0 || strcmp(argv[i], "-a") == 0) && argc > i+1)
        {
            anisotropy = atof(argv[i+1]);
            anisotropyProvided = true;
        }
        else if((strcmp(argv[i], "-speed") == 0 || strcmp(argv[i], "-s") == 0) && argc > i+1)
        {
            speed = atof(argv[i+1]);
            speedProvided = true;
        }
        else if((strcmp(argv[i], "-maxIterations") == 0 || strcmp(argv[i], "-m") == 0) && argc > i+1)
            maxIterations = atoi(argv[i+1]);
        else if((strcmp(argv[i], "-maxBlocks") == 0 || strcmp(argv[i], "-b") == 0) && argc > i+1)
            maxBlocks = atoi(argv[i+1]);
        else if((strcmp(argv[i], "-maxVertsPerBlock") == 0 || strcmp(argv[i], "-vb") == 0) && argc > i+1)
            maxVertsPerBlock = atoi(argv[i+1]);
        else if(strcmp(argv[i], "-verbose") == 0 || strcmp(argv[i], "-v") == 0)
            verbose = true;
    }
    
    if(inFile.empty())
    {
        std::cout << "ERROR:  Missing parameter -inFile or -i" << std::endl;
        return(EXIT_FAILURE);
    }
    if(seedsFile.empty())
    {
        std::cout << "ERROR:  Missing parameter -seedsFile or -p" << std::endl;
        return(EXIT_FAILURE);
    }
    if(outFile.empty())
    {
        std::cout << "ERROR:  Missing parameter -outFile or -o" << std::endl;
        return(EXIT_FAILURE);
    }
    
    std::cout << "INFO:   maxIterations is " << maxIterations << std::endl;
    std::cout << "INFO:   maxBlocks is " << maxBlocks << std::endl;
    std::cout << "INFO:   maxVertsPerBlock is " << maxVertsPerBlock << std::endl;
    
    std::ofstream outFileStream(outFile);
    if(outFileStream.is_open())
        outFileStream.close();
    else
    {
        std::cout << "ERROR:  Could not open output file" << std::endl;
        return(EXIT_FAILURE);
    }
    
    int pos = inFile.find_last_of(".");
    std::string extension = inFile.substr(pos+1, inFile.size()-pos);
    std::transform(extension.begin(), extension.end(), extension.begin(), ::tolower);
    
    std::cout << "STATUS: Loading input file " << inFile << "..." << std::endl;
    
    vtkSmartPointer<vtkUnstructuredGrid> uGrid;
    if(extension == "vtk")
    {
        vtkSmartPointer<vtkDataSetReader> vtkReader = vtkSmartPointer<vtkDataSetReader>::New();
        vtkReader->SetFileName(inFile.c_str());
        if(!vtkReader->OpenVTKFile())
        {
            std::cout << "ERROR:  VTK file could not be opened - does it exist?" << std::endl;
            return(EXIT_FAILURE);
        }
        vtkReader->Update();
        if(!vtkReader->IsFileUnstructuredGrid())
        {
            std::cout << "ERROR:  VTK file does not contain an unstructured grid" << std::endl;
            return(EXIT_FAILURE);
        }
        uGrid = vtkReader->GetUnstructuredGridOutput();
    }
    else if(extension == "vtu")
    {
        vtkSmartPointer<vtkXMLUnstructuredGridReader> vtuReader = vtkSmartPointer<vtkXMLUnstructuredGridReader>::New();
        if(!vtuReader->CanReadFile(inFile.c_str()))
        {
            std::cout << "ERROR:  VTU file could not be opened - does it exist?" << std::endl;
            return(EXIT_FAILURE);
        }
        vtuReader->SetFileName(inFile.c_str());
        vtuReader->Update();
        uGrid = vtuReader->GetOutput();
    }
    
    // create cell list
    int numCells = (int)uGrid->GetNumberOfCells();
    int* cells = new int[4*numCells];
    for(int i = 0; i < numCells; i++)
    {
        vtkCell* cell = uGrid->GetCell(i);
        if(cell->GetCellType() != VTK_TETRA)
        {
            std::cout << "ERROR:  Only linear tetrahedrons are supported" << std::endl;
            return(EXIT_FAILURE);
        }
        for(int j = 0; j < 4; j++)
            cells[4*i+j] = (int)cell->GetPointId(j);
    }
    
    // create point list
    int numPoints = uGrid->GetNumberOfPoints();
    float* points = new float[3*numPoints];
    for(int i = 0; i < numPoints; i++)
    {
        for(int j = 0; j < 3; j++)
            points[3*i+j] = (float)uGrid->GetPoint(i)[j];
    }
    
    std::cout << "STATUS: Loading seeds file " << seedsFile << "..." << std::endl;
    
    std::vector< std::vector<int> > seeds;
    std::ifstream seedsFileStream(seedsFile);
    if(!seedsFileStream.is_open())
    {
        std::cout << std::endl << "ERROR:  Could not open seeds file" << std::endl;
        return(EXIT_FAILURE);
    }
    std::string line;
    while(std::getline(seedsFileStream, line))
    {
        std::istringstream iss(line);
        std::vector<int> s;
        int i;
        while(iss >> i)
            s.push_back(i);
        if(s.size() > 0)
            seeds.push_back(s);
    }
    seedsFileStream.close();
    
    if(seeds.size() == 0)
    {
        std::cout << "ERROR:  No seed configuration found" << std::endl;
        return(EXIT_FAILURE);
    }
    std::cout << "INFO:   " << seeds.size() << " seed configuration(s) found" << std::endl;
    
    // create slowness values
    std::vector<float> slownessMtx;
    bool isotropicCase = true;
    vtkCellData* cellData = uGrid->GetCellData();
    
    std::vector<float> slownessVals(numCells, 1.0/(speed*speed));
    if(speedProvided)
        std::cout << "INFO:   Using global speed of " << speed << std::endl;
    else if(cellData->HasArray("Speed"))
    {
        vtkDataArray* speedArray = cellData->GetArray("Speed");
        for(int i = 0; i < numCells; i++)
        {
            float s = speedArray->GetComponent(i, 0);
            slownessVals[i] = 1.0/(s*s);
        }
        std::cout << "INFO:   Using local speeds from cell data array 'Speed'" << std::endl;
    }
    else
        std::cout << "INFO:   Using default global speed of " << speed << std::endl;
    
    std::vector<float> anisotropyVals;
    if(anisotropyProvided && anisotropy != 1.0)
    {
        anisotropyVals.resize(numCells);
        std::fill(anisotropyVals.begin(), anisotropyVals.end(), anisotropy);
        isotropicCase = false;
        std::cout << "INFO:   Anisotropic case" << std::endl;
        std::cout << "INFO:   Using global anisotropy ratio of " << anisotropy << std::endl;
    }
    else if(cellData->HasArray("Anisotropy"))
    {
        anisotropyVals.resize(numCells);
        vtkDataArray* anisotropyArray = cellData->GetArray("Anisotropy");
        for(int i = 0; i < numCells; i++)
            anisotropyVals[i] = anisotropyArray->GetComponent(i, 0);
        isotropicCase = false;
        std::cout << "INFO:   Anisotropic case" << std::endl;
        std::cout << "INFO:   Using local anisotropy ratios from cell data array 'Anisotropy'" << std::endl;
    }
    
    if(isotropicCase)
    {
        // isotropic case: only 1 value for each cell in slownessMtx
        slownessMtx = slownessVals;
        std::cout << "INFO:   Isotropic case" << std::endl;
    }
    else
    {
        // anisotropic case: 6 values for each cell in slownessMtx,
        // representing one triangular part of the symmetric matrix
        if(cellData->HasArray("Phi") && cellData->HasArray("Theta"))
        {
            slownessMtx.resize(6*numCells);
            vtkDataArray* phiArray = cellData->GetArray("Phi");
            vtkDataArray* thetaArray = cellData->GetArray("Theta");
            for(int i = 0; i < numCells; i++)
            {
                float phi = phiArray->GetComponent(i, 0);
                float theta = thetaArray->GetComponent(i, 0);
                Matrix3<float> M = calculateTensor(phi, theta, anisotropyVals[i]);
                slownessMtx[6*i+0] = slownessVals[i] * M(0);
                slownessMtx[6*i+1] = slownessVals[i] * M(1);
                slownessMtx[6*i+2] = slownessVals[i] * M(2);
                slownessMtx[6*i+3] = slownessVals[i] * M(4);
                slownessMtx[6*i+4] = slownessVals[i] * M(5);
                slownessMtx[6*i+5] = slownessVals[i] * M(8);
            }
            std::cout << "INFO:   Using cell data arrays 'Phi' and 'Theta' to define orientation of anisotropy" << std::endl;
        }
        else
        {
            std::cout << "ERROR:  Cell data arrays 'Phi' and 'Theta' are required for anisotropic case" << std::endl;
            return(EXIT_FAILURE);
        }
    }
    
    std::cout << "STATUS: Converting mesh..." << std::endl;
    
    TetMesh* tetMesh = new TetMesh();
    tetMesh->init(points, numPoints,
                  NULL, 0, // trilist, numtri
                  cells, numCells,
                  NULL, // attrlist
                  slownessMtx,
                  verbose);
    tetMesh->need_neighbors(verbose);
    tetMesh->need_adjacenttets(verbose);
    tetMesh->need_tet_virtual_tets(verbose);
    
    meshFIM3dEikonal* fim3d = new meshFIM3dEikonal;
    fim3d->SetMesh(tetMesh);
    
    std::cout << "STATUS: Partitioning..." << std::endl;
    
    fim3d->GraphPartition_METIS2(maxBlocks, maxVertsPerBlock, verbose);
    fim3d->m_numBlock = maxBlocks;
    fim3d->PartitionTets(maxBlocks, verbose);
    
    std::cout << "INFO:   maxNumTotalTets is " << fim3d->m_maxNumTotalTets << std::endl;
    
    // create a copy of fim3d, so that it can be restored for multiple runs
    meshFIM3dEikonal fim3d_copy = *fim3d;
    
    pos = outFile.find_last_of(".");
    extension = outFile.substr(pos+1, outFile.size()-pos);
    std::transform(extension.begin(), extension.end(), extension.begin(), ::tolower);
    bool binaryOutput = (extension == "bin") ? true : false;
    
    if(binaryOutput)
    {
        std::cout << "INFO:   Using binary output format" << std::endl;
        outFileStream.open(outFile, std::ofstream::out | std::ofstream::app | std::ofstream::binary);
    }
    else
    {
        std::cout << "INFO:   Using ascii output format" << std::endl;
        outFileStream.open(outFile, std::ofstream::out | std::ofstream::app);
    }
    if(!outFileStream.is_open())
    {
        std::cout << "ERROR:  Could not open output file" << std::endl;
        return(EXIT_FAILURE);
    }
    if(binaryOutput)
        outFileStream.write(reinterpret_cast<const char*>(&numPoints), sizeof(int));
    else
        outFileStream.precision(7);
    
    for(int i = 0; i < seeds.size(); i++)
    {
        std::cout << "STATUS: Solving - seed configuration " << i+1 << " of " << seeds.size() << "..." << std::endl;
        
        // restore fim3d
        *fim3d = fim3d_copy;
        
        fim3d->SetSeedPoint(seeds[i]);
        std::vector< std::vector<float> > iterVals;
        iterVals = fim3d->GenerateData(maxIterations, verbose);
        std::vector<float> vals = iterVals.back();
        
        if(binaryOutput)
        {
            for(int i = 0; i < vals.size(); i++)
                outFileStream.write(reinterpret_cast<const char*>(&vals[i]), sizeof(float));
        }
        else
        {
            for(int i = 0; i < vals.size(); i++)
                outFileStream << vals[i] << " ";
            outFileStream << std::endl;
        }
        
        //std::vector< std::vector<float> > exportVals;
        //exportVals.push_back(iterVals.back());
        //fim3d->writeVTK(exportVals);
    }
    
    outFileStream.close();
    
    return(EXIT_SUCCESS);
}
