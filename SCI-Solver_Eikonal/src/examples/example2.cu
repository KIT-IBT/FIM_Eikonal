#include "hip/hip_runtime.h"
//-------------------------------------------------------------------
//
//  Copyright (C) 2015
//  Scientific Computing & Imaging Institute
//  University of Utah
//
//  Permission is  hereby  granted, free  of charge, to any person
//  obtaining a copy of this software and associated documentation
//  files  ( the "Software" ),  to  deal in  the  Software without
//  restriction, including  without limitation the rights to  use,
//  copy, modify,  merge, publish, distribute, sublicense,  and/or
//  sell copies of the Software, and to permit persons to whom the
//  Software is  furnished  to do  so,  subject  to  the following
//  conditions:
//
//  The above  copyright notice  and  this permission notice shall
//  be included  in  all copies  or  substantial  portions  of the
//  Software.
//
//  THE SOFTWARE IS  PROVIDED  "AS IS",  WITHOUT  WARRANTY  OF ANY
//  KIND,  EXPRESS OR IMPLIED, INCLUDING  BUT NOT  LIMITED  TO THE
//  WARRANTIES   OF  MERCHANTABILITY,  FITNESS  FOR  A  PARTICULAR
//  PURPOSE AND NONINFRINGEMENT. IN NO EVENT  SHALL THE AUTHORS OR
//  COPYRIGHT HOLDERS  BE  LIABLE FOR  ANY CLAIM, DAMAGES OR OTHER
//  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
//  ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE
//  USE OR OTHER DEALINGS IN THE SOFTWARE.
//-------------------------------------------------------------------
//-------------------------------------------------------------------
#include <Eikonal.h>

int main(int argc, char* argv[]) {
  Eikonal data(true);
  for (int i = 1; i < argc; i++) {
    if (strcmp(argv[i], "-v") == 0) {
      data.verbose_ = true;
    } else if (strcmp(argv[i], "-i") == 0) {
      if (i + 1 >= argc) break;
      data.filename_ = std::string(argv[i + 1]);
      i++;
    } else if (strcmp(argv[i], "-s") == 0) {
      if (i + 1 >= argc) break;
      std::string type = std::string(argv[i + 1]);
      if (type == "CURVATURE") {
        data.speedType_ = CURVATURE;
      } else if (type == "NOISE") {
        data.speedType_ = NOISE;
      } else if (type == "MANUAL"){
        data.speedType_ = MANUAL_SPEED;
      }
      i++;
    } else if (strcmp(argv[i], "-x") == 0) {
      while (i + 1 < argc && argv[i + 1][0] != '-') {
        std::ifstream mat(argv[++i]);
        while (mat.good()) {
          float val;
          mat >> val;
          if (!mat.good()) break;
          data.speedMtxMultipliers_.push_back(val);
        }
        mat.close();
      }
    } else if (strcmp(argv[i], "-b") == 0) {
      if (i + 1 >= argc) break;
      data.maxVertsPerBlock_ = atoi(argv[i + 1]);
      i++;
    } else if (strcmp(argv[i], "-n") == 0) {
      if (i + 1 >= argc) break;
      data.maxIterations_= atoi(argv[i + 1]);
      i++;
    } else if (strcmp(argv[i], "-h") == 0) {
      printf("Usage: ./Example2 [OPTIONS]\n");
      printf("  -h              Show this help.\n");
      printf("  -v              Verbose output.\n");
      printf("  -i INPUT        Use this triangle mesh \n");
      //# of verts/block affects partitioning & convergence.
      //Adjust accordingly.
      printf("  -b MAX_BLK_VERT Max # of verts/block to use\n");
      printf("  -n MAX_ITER     Max # of iterations to run\n");
      printf("  -s SPEEDTYPE    Speed type is [ONE], CURVATURE, or NOISE.\n");
      printf("  -x SCALAR_FILE  File with speed scalars per face.\n");
      exit(0);
    }
  }
  data.initializeMesh();
  std::ofstream out("square_scalars.txt");
  for (size_t i = 0; i < data.triMesh_->faces.size(); i++) {
    point p = (data.triMesh_->vertices[data.triMesh_->faces[i][0]] +
      data.triMesh_->vertices[data.triMesh_->faces[i][1]] +
      data.triMesh_->vertices[data.triMesh_->faces[i][2]]) / 3.f;
    if (len(p - point(16., 16., 0.)) < 6.) {
      out << 20. << std::endl;
    } else {
      out << 1. << std::endl;
    }
  }
  out.close(); 

  data.solveEikonal();
  //write the output to file
  data.writeVTK(false);
  //the solution for the sphere examples (center 0,0,0, & radius 100)
  std::vector< float > solution;
  solution.resize(data.triMesh_->vertices.size());
  point first = data.triMesh_->vertices[0];
  for (size_t i = 0; i < solution.size(); i++) {
    float xDot = data.triMesh_->vertices[i][0];
    float yDot = data.triMesh_->vertices[i][1];
    float zDot = data.triMesh_->vertices[i][2];
    solution[i] = 100.f * std::acos(
      (first[0] * xDot + first[1] * yDot + first[2] * zDot) /
      std::sqrt(xDot * xDot + yDot * yDot + zDot * zDot) /
      len(first));
  }
  if (data.verbose_)
    data.printErrorGraph(solution);
  return 0;
}
