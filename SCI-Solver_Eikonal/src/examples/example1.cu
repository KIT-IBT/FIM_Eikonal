#include "hip/hip_runtime.h"
//-------------------------------------------------------------------
//
//  Copyright (C) 2015
//  Scientific Computing & Imaging Institute
//  University of Utah
//
//  Permission is  hereby  granted, free  of charge, to any person
//  obtaining a copy of this software and associated documentation
//  files  ( the "Software" ),  to  deal in  the  Software without
//  restriction, including  without limitation the rights to  use,
//  copy, modify,  merge, publish, distribute, sublicense,  and/or
//  sell copies of the Software, and to permit persons to whom the
//  Software is  furnished  to do  so,  subject  to  the following
//  conditions:
//
//  The above  copyright notice  and  this permission notice shall
//  be included  in  all copies  or  substantial  portions  of the
//  Software.
//
//  THE SOFTWARE IS  PROVIDED  "AS IS",  WITHOUT  WARRANTY  OF ANY
//  KIND,  EXPRESS OR IMPLIED, INCLUDING  BUT NOT  LIMITED  TO THE
//  WARRANTIES   OF  MERCHANTABILITY,  FITNESS  FOR  A  PARTICULAR
//  PURPOSE AND NONINFRINGEMENT. IN NO EVENT  SHALL THE AUTHORS OR
//  COPYRIGHT HOLDERS  BE  LIABLE FOR  ANY CLAIM, DAMAGES OR OTHER
//  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
//  ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE
//  USE OR OTHER DEALINGS IN THE SOFTWARE.
//-------------------------------------------------------------------
//-------------------------------------------------------------------

#include <Eikonal.h>

int main(int argc, char *argv[])
{
  //3D options
  Eikonal data(false);
  //input filename (minus extension)
  data.filename_ = "../src/test/test_data/sphere334";
  for (int i = 0; i < argc; i++)
    if (strcmp(argv[i], "-v") == 0) {
    data.verbose_ = true;
    } else if (strcmp(argv[i], "-m") == 0) {
    if (i + 1 >= argc) break;
    data.maxIterations_ = atoi(argv[i + 1]);
    i++;
    } else if (strcmp(argv[i], "-b") == 0) {
      if (i + 1 >= argc) break;
      data.maxBlocks_ = atoi(argv[i + 1]);
      i++;
    } else if (strcmp(argv[i], "-x") == 0) {
      while (i + 1 < argc && argv[i + 1][0] != '-') {
        std::ifstream mat(argv[++i]);
        while (mat.good()) {
          float val;
          mat >> val;
          if (!mat.good()) break;
          data.speedMtxMultipliers_.push_back(val);
        }
        mat.close();
      }
    } else if (strcmp(argv[i], "-i") == 0) {
      if (i + 1 >= argc) break;
      data.filename_ = std::string(argv[i + 1]);
      i++;
    } else if (strcmp(argv[i], "-h") == 0) {
      printf("Usage: ./Example1 [OPTIONS]\n");
      printf("  -h              Show this help.\n");
      printf("  -v              Verbose output.\n");
      printf("  -i INPUT        Use this tet mesh \n");
      printf("  -b MAX_BLOCKS   Max # of blocks to use\n");
      printf("  -m MAX_ITER     Max # of iterations before quit\n");
      // The tensors are 6 unique values in a matrix per tet
      // [ 0 1 2 ]
      // [ 1 3 4 ]
      // [ 2 4 5 ]
      printf("  -x MATRIX_FILE  File of tensor matrices per tet (N*6 floats).\n");
      printf("                  N floats if considered scalar speeds per tet.\n");
      exit(0);
    }
    data.solveEikonal();
    //write out the VTK files
    data.writeVTK(false); //true to output values at each iter.
    //we know that the solution should be the euclidean distance from the center.
    std::vector <float> solution;
    for (size_t i = 0; i < data.tetMesh_->vertices.size(); i++) {
      float x = data.tetMesh_->vertices[i][0];
      float y = data.tetMesh_->vertices[i][1];
      float z = data.tetMesh_->vertices[i][2];
      solution.push_back(std::sqrt((0.f - x)*(0.f - x) + (0.f - y)*(0.f - y) + (0.f - z)*(0.f - z)));
    }
    if (data.verbose_)
      data.printErrorGraph(solution);
    return 0;
}

