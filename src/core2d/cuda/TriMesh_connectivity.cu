#include "hip/hip_runtime.h"
/*
   Szymon Rusinkiewicz
   Princeton University

   TriMesh_connectivity.cc
   Manipulate data structures that describe connectivity between faces and verts.
 */


#include <stdio.h>
#include <time.h>
#include "TriMesh.h"
#include <algorithm>

void TriMesh::need_Rinscribe()
{
  need_faceedges();
  int nf = faces.size();
  if (!radiusInscribe.empty())
  {
    return;
  }

  radiusInscribe.resize(nf);

  for (int i=0; i<nf; i++)
  {
    Face f = faces[i];
    double e1 = f.edgeLens[0];
    double e2 = f.edgeLens[1];
    double e3 = f.edgeLens[2];
    double s = (e1+e2+e3)/2.0;
    radiusInscribe[i] = sqrt(s *(s-e1) * (s-e2) * (s - e3)) / s;
  }

}


void TriMesh::need_faces()
{
  if (!faces.empty())
  {
    return;
  }
  if (!tstrips.empty())
    unpack_tstrips();
  else if (!grid.empty())
    triangulate_grid();


}


void TriMesh::need_faceedges()
{
  if (faces.empty())
  {
    printf("No faces to compute face edges!!!\n");
    return;
  }
  int numFaces = faces.size();
  for (int i = 0; i < numFaces; i++)
  {
    Face f = faces[i];
    point edge01 = vertices[f[1]] - vertices[f[0]];
    point edge12 = vertices[f[2]] - vertices[f[1]];
    point edge20 = vertices[f[0]] - vertices[f[2]];
    faces[i].edgeLens[0] =sqrt(edge01[0]*edge01[0] + edge01[1]*edge01[1] + edge01[2]*edge01[2]);
    faces[i].edgeLens[1] =sqrt(edge12[0]*edge12[0] + edge12[1]*edge12[1] + edge12[2]*edge12[2]);
    faces[i].edgeLens[2] =sqrt(edge20[0]*edge20[0] + edge20[1]*edge20[1] + edge20[2]*edge20[2]);


  }

}

void TriMesh::need_noise(float low, float high)
{
  noiseOnVert.clear();
  need_neighbors();
  int nv = vertices.size();
  noiseOnVert.resize(nv);
  srand( (unsigned)time( NULL ) );

  for (int i = 0;i<nv; i++) {
    noiseOnVert[i] = (float)rand() /
      (RAND_MAX)*(high - low) + low;  //random number between [low,high]
  }

  //iterate
  int iterNum = 0;
  for (int i=0;i<iterNum; i++) {
    for (int j=0; j<nv;j++) {
      noiseOnVert[j] = 0;
      std::vector<int> nb = neighbors[j];
      for (int k=0;k<nb.size();k++) {
        noiseOnVert[j] +=noiseOnVert[neighbors[j][k]];
      }
      noiseOnVert[j] /= nb.size();
    }
  }
}

void TriMesh::need_speed()
{
  //this is a set of optional speed functions to use.
  int nf = faces.size();

  for (int i =0; i<nf;i++)
  {
    switch (this->speed_type_)
    {
    case CURVATURE:
      faces[i].speedInv = (abs(curv1[faces[i][0]] +
            curv2[faces[i][0]]) +
          std::abs(curv1[faces[i][1]] + curv2[faces[i][1]]) +
          std::abs(curv1[faces[i][2]] +
            curv2[faces[i][2]])) / 6.0;
      break;
    case ONE:
      faces[i].speedInv = 1.0;
      break;
    case NOISE:
      faces[i].speedInv =( noiseOnVert[faces[i][0]] +
          noiseOnVert[faces[i][1]] +
          noiseOnVert[faces[i][2]] ) / 3;
      break;
    default:
      faces[i].speedInv = 1.0;
      break;
    }
  }
}


// Find the direct neighbors of each vertex
void TriMesh::need_neighbors(bool verbose)
{
  if (!neighbors.empty())
    return;
  need_faces();

  if (verbose)
    printf("Finding vertex neighbors... ");
  int nv = vertices.size(), nf = faces.size();

  std::vector<int> numneighbors(nv);
  for (int i = 0; i < nf; i++) {
    numneighbors[faces[i][0]]++;
    numneighbors[faces[i][1]]++;
    numneighbors[faces[i][2]]++;
  }

  neighbors.resize(nv);
  for (int i = 0; i < nv; i++)
    neighbors[i].reserve(numneighbors[i]+2); // Slop for boundaries

  for (int i = 0; i < nf; i++) {
    for (int j = 0; j < 3; j++) {
      std::vector<int> &me = neighbors[faces[i][j]];
      int n1 = faces[i][(j+1)%3];
      int n2 = faces[i][(j+2)%3];
      if (find(me.begin(), me.end(), n1) == me.end())
        me.push_back(n1);
      if (find(me.begin(), me.end(), n2) == me.end())
        me.push_back(n2);
    }
  }

  if(verbose)
    printf("Done.\n");
}


// Find the faces touching each vertex
void TriMesh::need_adjacentfaces(bool verbose)
{
  if (!adjacentfaces.empty())
    return;
  need_faces();

  if (verbose)
    printf("Finding vertex to triangle maps... ");
  int nv = vertices.size(), nf = faces.size();

  std::vector<int> numadjacentfaces(nv);
  for (int i = 0; i < nf; i++) {
    numadjacentfaces[faces[i][0]]++;
    numadjacentfaces[faces[i][1]]++;
    numadjacentfaces[faces[i][2]]++;
  }

  adjacentfaces.resize(vertices.size());
  for (int i = 0; i < nv; i++)
    adjacentfaces[i].reserve(numadjacentfaces[i]);

  for (int i = 0; i < nf; i++) {
    for (int j = 0; j < 3; j++)
      adjacentfaces[faces[i][j]].push_back(i);
  }

  if(verbose)
    printf("Done.\n");
}

void TriMesh::need_face_virtual_faces()
{

  std::vector<Face> t_faces;
  Face f;
  int numFaces = faces.size();
  faceVirtualFaces.resize(numFaces);
  for (int i = 0; i < numFaces; i++)
  {
    t_faces.clear();
    f = faces[i];

    for (int j = 0; j< 3 ; j++)
    {
      if(!IsNonObtuse(f[j],f))// check angle: if non-obtuse, return existing face
      {
        int nfae = across_edge[i][j];
        if (nfae > -1)
        {
          SplitFace(t_faces,f[j],f,nfae);// if obtuse, split face till we get all acute angles
        }
        else
          printf("NO cross edge!!! Maybe a hole!!\n");

      }
    }

    faceVirtualFaces[i] = t_faces;
  }
}


// Find the face across each edge from each other face (-1 on boundary)
// If topology is bad, not necessarily what one would expect...
void TriMesh::need_across_edge()
{
  if (!across_edge.empty())
    return;
  need_adjacentfaces();

  dprintf("Finding across-edge maps... ");

  int nf = faces.size();
  across_edge.resize(nf, Face(-1,-1,-1));

  for (int i = 0; i < nf; i++) {
    for (int j = 0; j < 3; j++) {
      if (across_edge[i][j] != -1)
        continue;
      int v1 = faces[i][(j+1)%3];
      int v2 = faces[i][(j+2)%3];
      const std::vector<int> &a1 = adjacentfaces[v1];
      const std::vector<int> &a2 = adjacentfaces[v2];
      for (int k1 = 0; k1 < a1.size(); k1++) {
        int other = a1[k1];
        if (other == i)
          continue;
        std::vector<int>::const_iterator it =
          std::find(a2.begin(), a2.end(), other);
        if (it == a2.end())
          continue;
        int ind = (faces[other].indexof(v1)+1)%3;
        if (faces[other][(ind+1)%3] != v2)
          continue;
        across_edge[i][j] = other;
        across_edge[other][ind] = i;
        break;
      }
    }
  }

  dprintf("Done.\n");
}

